#include "hip/hip_runtime.h"
﻿#include ""
#include "hip/hip_runtime.h"

#include <iostream>

#include "gpu_image_processor.cuh"

#define DELLEXPORT extern "C" __declspec(dllexport)


__forceinline__ __device__ void DeviceImageProcessor::getWindow(float* R, float* G, float* B, const float* img, int Row, int Col, size_t X) {
    const int n_pixels = 9;
    const int window_pos = n_pixels / 9;
    unsigned int index = 0;
    unsigned c = 0;
    for (int i = -window_pos; i <= window_pos; i++) {
        for (int j = -window_pos; j <= window_pos; j++) {
            index = (Row + i) * X + (Col + j);
            R[c] = static_cast<float>(img[index * 3 + 0]);
            G[c] = static_cast<float>(img[index * 3 + 1]);
            B[c] = static_cast<float>(img[index * 3 + 2]);
            c++;
        }
    }
}


__forceinline__ __device__ float DeviceImageProcessor::getL1(float r1, float r2, float g1, float g2, float b1, float b2) {
    return fabsf(r1 - r2) + fabsf(g1 - g2) + fabsf(b1 - b2);
}


__forceinline__ __device__ void DeviceImageProcessor::getAlphaVmf(float* vectR, float* vectG, float* vectB, float* alpha_values, const unsigned int n_pixels) {
    float alpha = 0;

    for (unsigned int a = 0; a < n_pixels; a++) {
        for (unsigned int b = 0; b < n_pixels; b++) {
            alpha += getL1(vectR[a], vectR[b], vectG[a], vectG[b], vectB[a], vectB[b]);
        }

        alpha_values[a] = alpha;
        alpha = 0;
    }
}
__forceinline__ __device__ void DeviceImageProcessor::selectionSort(int* positions, const float* alphaValues, int n) {
    for (int i = 0; i < n - 1; ++i) {
        int minIdx = i;
        for (int j = i + 1; j < n; ++j) {
            if (alphaValues[positions[j]] < alphaValues[positions[minIdx]]) {
                minIdx = j;
            }
        }
        // Swap positions[i] and positions[minIdx]
        int temp = positions[i];
        positions[i] = positions[minIdx];
        positions[minIdx] = temp;
    }
}


__global__ void vmf_gpu(float* out, const float* in, size_t Y, size_t X) {
    int Row = blockIdx.y * blockDim.y + threadIdx.y;
    int Col = blockIdx.x * blockDim.x + threadIdx.x;

    DeviceImageProcessor processor;

    float vectR[9], vectG[9], vectB[9];
    float alphas[9];
    int positions[9];

    const unsigned int n_pixels = 9;
    unsigned int output_pixel_index = 0;

    if ((Row > 1) && (Col > 1) && (Row < Y - 1) && (Col < X - 1)) {
        processor.getWindow(vectR, vectG, vectB, in, Row, Col, X);
        processor.getAlphaVmf(vectR, vectG, vectB, alphas, n_pixels);

        // Necessary for selectionSort
        for (int i = 0; i < n_pixels; ++i) {
            positions[i] = i;
        }
        processor.selectionSort(positions, alphas, n_pixels);

        unsigned int output_pixel_index = positions[0];

        // Set the output pixel values
        out[(Row * X + Col) * 3 + 0] = static_cast<unsigned char>(vectR[output_pixel_index]);
        out[(Row * X + Col) * 3 + 1] = static_cast<unsigned char>(vectG[output_pixel_index]);
        out[(Row * X + Col) * 3 + 2] = static_cast<unsigned char>(vectB[output_pixel_index]);
    }
}


DELLEXPORT void run_gpu_filter(float* img_filtered, const float* img_noisy, size_t Y, size_t X) {

    hipError_t cudaStatus;
    int device_count = 0;
    cudaStatus = hipGetDeviceCount(&device_count);
    if (cudaStatus != hipSuccess || device_count == 0) {
        goto Error;
    }

    float* device_img_noisy = nullptr;
    float* device_img_filtered = nullptr;

    size_t img_size = Y * X * sizeof(float) * 3;

    cudaStatus = hipMalloc((void**)&device_img_noisy, img_size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed!" << std::endl;
        goto Error;
    }
    cudaStatus = hipMalloc((void**)&device_img_filtered, img_size);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMalloc failed!" << std::endl;
        goto Error;
    }
    cudaStatus = hipMemcpy(device_img_noisy, img_noisy, img_size, hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed!" << std::endl;
        goto Error;
    }

    int nHilosporBloque = 8;
    dim3 nThreads(nHilosporBloque, nHilosporBloque, 1);
    dim3 nBloques((X / nHilosporBloque) + 1, (Y / nHilosporBloque) + 1, 1);


    vmf_gpu << <nBloques, nThreads >> > (device_img_filtered, device_img_noisy, Y, X);


    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        std::cerr << "Kernel launch failed: " << hipGetErrorString(cudaStatus) << std::endl;
        goto Error;
    }
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipDeviceSynchronize returned error code " << cudaStatus << " after launching kernel!" << std::endl;
        goto Error;
    }
    cudaStatus = hipMemcpy(img_filtered, device_img_filtered, img_size, hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        std::cerr << "hipMemcpy failed!" << std::endl;
        goto Error;
    }

    hipFree(device_img_noisy);
    hipFree(device_img_filtered);


Error:
    hipFree(device_img_noisy);
    hipFree(device_img_filtered);
}
